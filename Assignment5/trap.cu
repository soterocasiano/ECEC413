/* ld and execute as follows:
        make clean && make
        ./trap a b n

 * Author: Naga Kandasamy
 * Date modified: February 28, 2025

 * Student name(s): Sotero Casiano, Jeffrey Lau
 * Date modified: 3/5/2025
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>

/* Include the kernel code */
#include "trap_kernel.cu"

double compute_on_device(float, float, int, float);
extern "C" double compute_gold(float, float, int, float);

double get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec * 1e-6;
}

int main(int argc, char **argv)
{
    if (argc < 4) {
        fprintf(stderr, "Usage: %s a b n\n", argv[0]);
        fprintf(stderr, "a: Start limit. \n");
        fprintf(stderr, "b: end limit\n");
        fprintf(stderr, "n: number of trapezoids\n");
        exit(EXIT_FAILURE);
    }

    int a = atoi(argv[1]); /* Left limit */
    int b = atoi(argv[2]); /* Right limit */
    int n = atoi(argv[3]); /* Number of trapezoids */

    float h = (b-a)/(float)n; // Height of each trapezoid
        printf("Number of trapezoids = %d\n", n);
    printf("Height of each trapezoid = %f \n", h);
        double cpu_start_time = get_time();
        double reference = compute_gold(a, b, n, h);
        double cpu_end_time = get_time();
        printf("Reference solution computed on the CPU = %f \n", reference);
        printf("CPU time = %f seconds\n", cpu_end_time - cpu_start_time);
        /* Write this function to complete the trapezoidal on the GPU. */
        double gpu_start_time = get_time();
        double gpu_result = compute_on_device(a, b, n, h);
        double gpu_end_time = get_time();
        printf("Solution computed on the GPU = %f \n", gpu_result);
        printf("GPU time = %f seconds\n", gpu_end_time - gpu_start_time);
}

/* Complete this function to perform the trapezoidal rule on the GPU. */
double compute_on_device(float a, float b, int n, float h)
{
    float *d_global_sum;
    float h_global_sum = 0.0f;

    // Allocate memory on the GPU
    hipMalloc((void **)&d_global_sum, sizeof(float));
    hipMemcpy(d_global_sum, &h_global_sum, sizeof(float), hipMemcpyHostToDevice);

    // Define thread block and grid sizes
    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    // Launch the kernel
    trap_kernel<<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>(a, b, n, h, d_global_sum);
    hipDeviceSynchronize();
    // Copy the result back to the host
    hipMemcpy(&h_global_sum, d_global_sum, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_global_sum);

    return (double)h_global_sum;
}
